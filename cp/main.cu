#include <iostream>
#include <vector>
#include <math.h>
#include <iomanip>
#include "hip/hip_runtime.h"


const double eps = 1e-7;

#define CSC(call)                                                   \
do {                                                                \
    hipError_t res = call;                                         \
    if (res != hipSuccess) {                                       \
        fprintf(stderr, "ERROR in %s:%d. Message: %s\n",            \
                __FILE__, __LINE__, hipGetErrorString(res));       \
        exit(0);                                                    \
    }                                                               \
} while(0)

__device__ void atomic_max(double* const address, const double value)
{
    unsigned long long int* const _address = (unsigned long long int*)address;
    unsigned long long int prev = *_address, next;
    do {
        next = prev;
        if (__longlong_as_double(next) >= value) {
            break;
        }
        prev = atomicCAS(_address, next, __double_as_longlong(value));
    } while (next != prev);
}

__global__ void reduce_max_col(double *matrix, int sz, double *value, int *idx) {
    __shared__ double sh_val;
    __shared__ int sh_idx;
    if (threadIdx.x == 0) {
        sh_val = 0.0;
        sh_idx = 0;
    }
    __syncthreads();

    double max_val = 0.0;
    int max_idx = 0;
    for (int i = threadIdx.x; i < sz; i += blockDim.x) {
        if (fabs(max_val) < fabs(matrix[i])) {
            max_val = matrix[i];
            max_idx = i;
        }
    }

    atomic_max(&sh_val, max_val);
    __syncthreads();
    if (sh_val == max_val) {
        sh_idx = max_idx;
    }

    __syncthreads();
    if (threadIdx.x == 0) {
        *value = sh_val;
        *idx = sh_idx;
    }
}

__global__ void swap_rows(double *matrix, int n, int m, int row1, int row2) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = blockDim.x * gridDim.x;

    while (j < m) {
        int idx1 = j * n + row1;
        int idx2 = j * n + row2;

        double tmp = matrix[idx1];
        matrix[idx1] = matrix[idx2];
        matrix[idx2] = tmp;

        j += offset;
    }
}

__global__ void gauss(double *matrix, int n, int m, int i) { 
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int offsetx = blockDim.x * gridDim.x;
    int offsety = blockDim.y * gridDim.y;

    int h = m - i - 1;
    int w = n - i - 1;

    for (int y = idy; y < h; y += offsety) {
        for (int x = idx; x < w; x += offsetx) {
            int k = i + 1 + y;
            int r = i + 1 + x;

            double coef = matrix[k * n + i];
            double num  = matrix[i * n + r];
            double div  = matrix[i * n + i];
            matrix[k * n + r] -= coef * num / div;
        }
    }
}

int main(int argc, char const *argv[])
{
    int n = 0, m = 0;
    scanf("%d %d", &n, &m);
    double *matrix = new double[n * (m + 1)];

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            std::cin >> matrix[j * n + i];
        }
    }
    for (int i = 0; i < n; i++) {
        std::cin >> matrix[m * n + i];
    }

    double *dev_matrix;
    CSC(hipMalloc(&dev_matrix, sizeof(double) * n * (m + 1)));
    CSC(hipMemcpy(dev_matrix, matrix, sizeof(double) * n * (m + 1), hipMemcpyHostToDevice));

    double *max_elem;
    int    *max_ptr;
    CSC(hipMalloc(&max_elem, sizeof(double)));
    CSC(hipMalloc(&max_ptr,  sizeof(int)));

    std::vector<bool> rank_cols(m, false);

    int skip = 0;
    int iterCount = std::min(n, m);
    for (int i = 0; i < iterCount; i++) {
        int begin_col_idx = (i + skip) * n + (i - skip);
        int end_col_idx   = (i + 1 + skip) * n;
        int col_size      = end_col_idx - begin_col_idx;

        reduce_max_col<<<512, 512>>>(dev_matrix + begin_col_idx, col_size, max_elem, max_ptr);
        
        double h_max_elem;
        int    h_max_ptr;
        CSC(hipMemcpy(&h_max_elem, max_elem, sizeof(double), hipMemcpyDeviceToHost));
        CSC(hipMemcpy(&h_max_ptr, max_ptr,   sizeof(int),    hipMemcpyDeviceToHost));

        int shift  = begin_col_idx + h_max_ptr;
        int center = (i + skip) * n + (i - skip);

        int row1 = shift  % n;
        int row2 = center % n;
        if (row1 != row2) {
            swap_rows<<<1024, 1024>>>(dev_matrix, n, m + 1, row1, row2);
            CSC(hipGetLastError());
            CSC(hipDeviceSynchronize());
        }

        double main_elem;
        CSC(hipMemcpy(&main_elem, dev_matrix + center, sizeof(double), hipMemcpyDeviceToHost));

        if (fabs(main_elem) < eps) {
            skip++;
            continue;
        } else {
            rank_cols[i + skip] = true;
        }

        gauss<<<dim3(64, 64), dim3(16, 16)>>>(dev_matrix, n, m + 1, i);
        CSC(hipGetLastError());
        CSC(hipDeviceSynchronize());
    }
    
    CSC(hipMemcpy(matrix, dev_matrix, sizeof(double) * n * (m + 1), hipMemcpyDeviceToHost));

    int rank = 0;
    for (int j = 0; j < m; j++) {
        if (rank_cols[j]) {
            rank++;
        }
    }
    std::cout << rank << "\n";

    if (rank == m) {
        std::cout << "System has one solution:\n";
    } else {
        std::cout << "System has infinity solutions. One of them:\n";
    }

    double *square_matrix = new double[rank * (rank + 1)];

    int ri = 0;
    for (int row = 0; row < n; row++) {
        if (row >= m) {
            break;
        }
        if (!rank_cols[row]) {
            continue;
        }

        int rj = 0;
        for (int j = 0; j < m; j++) {
            if (rank_cols[j]) {
                square_matrix[rj * rank + ri] = matrix[j * n + row];
                rj++;
            }
        }
        square_matrix[rank * rank + ri] = matrix[m * n + row];

        ri++;
        if (ri == rank) {
            break;
        }
    }

    for (int y = rank - 1; y >= 0; y--) {
        double diag = square_matrix[y * rank + y];
        if (fabs(diag) > eps) {
            for (int x = y - 1; x >= 0; x--) {
                double f = square_matrix[y * rank + x];
                square_matrix[rank * rank + x] -= square_matrix[rank * rank + y] * f / diag;
                square_matrix[y * rank + x] = 0.0;
            }
        }
    }

    std::cout << std::scientific << std::setprecision(10);
    for (int i = 0; i < rank; i++) {
        double diag = square_matrix[i * rank + i];
        double rhs  = square_matrix[rank * rank + i];
        double val  = (fabs(diag) < eps ? 0.0 : (rhs / diag));
        std::cout << val << " ";
    }
    for (int i = rank; i < m; i++) {
        std::cout << 0.0 << " ";
    }
    std::cout << std::endl;

    CSC(hipFree(dev_matrix));
    CSC(hipFree(max_elem));
    CSC(hipFree(max_ptr));

    delete[] square_matrix;
    delete[] matrix;

    return 0;
}
